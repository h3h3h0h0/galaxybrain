#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"
#include ""
#include "nbody.cuh"
#include <stdio.h>
#include <opencv2/opencv.hpp> 
#include <random>
#include <cmath>
#include <malloc.h>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <stdlib.h>

using namespace std;
using namespace cv;

int main()
{
    double* x = new double[5000];
    double* y = new double[5000];
    double* z = new double[5000];
    double* px = new double[5000];
    double* py = new double[5000];
    double* pz = new double[5000];
    const long double rad = 10000.0;
    uniform_real_distribution<double> unif1(-rad/2.0, rad / 2.0);
    normal_distribution<double> norm1(0, 1000);
    uniform_real_distribution<double> unif2(-10, 10);
    std::random_device                  rand_dev;
    std::mt19937                        generator(rand_dev());
    vector<Body> bodies;
    bodies.emplace_back(Body{ 0, 0, 0, 0, 0, 0, 100000000000.0 });

    Mat img = Mat::zeros(1000, 1000, CV_8UC1);
    for (int i = 0; i < 4999; i++) {
        double cx = unif1(generator);
        double cy = max(-5000.0, min(5000.0, norm1(generator)));
        if (unif2(generator) < 0) swap(cx, cy);
        double cz = unif2(generator);
        double m = 10000;
        if (i > 500) {
            m = 100;
            cx = unif1(generator);
            cy = unif1(generator);
        }
        double cr = sqrt(cx * cx + cy * cy + cz * cz);
        double cv = sqrt(6.674e-11 * 100000000000 / cr);
        double ca = atan2(cy, cx)+M_PI_2;
        double vx = cv * cos(ca);
        double vy = cv * sin(ca);
        double vz = 0;
        bodies.emplace_back(Body{ cx, cy, cz, vx, vy, vz, m});
    }
    NBody* bn = NULL;
    hipError_t err = NBody::makeNBody(bn, bodies, rad);
    Mat pimg = img.clone();
    for (int i = 0; i < 1000000000; i++) {
        img = Mat::zeros(1000, 1000, CV_8UC1);
        err = bn->step(1000, 10, 1024, 0);
        memcpy(&px, &x, sizeof(x));
        memcpy(&py, &y, sizeof(x));
        memcpy(&pz, &z, sizeof(x));
        err = bn->getPos(x, y, z);
        /*for (int j = 0; j < 100; j++) {
            cout<<"j: "<<j<<" dx: "<<x[j]-px[j]<<" dy: "<<y[j] - py[j] <<" dz: "<<z[j] - pz[j] <<endl;
        }*/
        //cout << "Error Pos: " << hipGetErrorString(err) << endl;
        for (int j = 0; j < 5000; j++) {
            int imgx = (int)(x[j] / rad * 500 + 500);
            imgx = min(1000, max(0, imgx));
            int imgy = (int)(y[j] / rad * 500 + 500);
            imgy = min(1000, max(0, imgy));
            img.at<uchar>(imgx, imgy) = (bodies[j].m == 100? 25 : (bodies[j].m == 10000? 70 : 255));
        }
        Mat oimg = Mat::ones(1000, 1000, CV_8UC1);
        oimg.setTo(25);
        Mat fimg = img + pimg - oimg;
        pimg = fimg.clone();
        putText(fimg, "Frame: " + to_string(i), Point(10, 30), FONT_HERSHEY_SIMPLEX, 1, Scalar(255, 255, 255), 2);
        imshow("Test", fimg);
        waitKey(1);
	}

    delete[] x;
    delete[] y;
    delete[] z;
    delete[] px;
    delete[] py;
    delete[] pz;

    return 0;
}