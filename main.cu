#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"
#include ""
#include "nbody.cuh"
#include <stdio.h>
#include <opencv2/opencv.hpp> 
#include <random>
#include <cmath>
#include <malloc.h>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <stdlib.h>
#include "render.cuh"

using namespace std;
using namespace cv;

int main()
{
    double* x = new double[5000];
    double* y = new double[5000];
    double* z = new double[5000];
    double* px = new double[5000];
    double* py = new double[5000];
    double* pz = new double[5000];
    const long double rad = 10000.0;
    uniform_real_distribution<double> unif1(-rad/2.0, rad / 2.0);
    normal_distribution<double> norm1(0, 1000);
    uniform_real_distribution<double> unif2(-10, 10);
    std::random_device                  rand_dev;
    std::mt19937                        generator(rand_dev());
    vector<Body> bodies;
    bodies.emplace_back(Body{ 0, 0, 0, 0, 0, 0, 100000000000.0, make_float3(0, 0, 0), make_float3(0, 0, 0), make_float3(0, 0, 0), 0, make_float2(0, 0), 1});

    Mat img = Mat::zeros(1000, 1000, CV_8UC1);
    for (int i = 0; i < 4999; i++) {
        double cx = unif1(generator);
        double cy = max(-5000.0, min(5000.0, norm1(generator)));
        if (unif2(generator) < 0) swap(cx, cy);
        double cz = unif2(generator);
        double m = 10000;
        if (i > 500) {
            m = 100;
            cx = unif1(generator);
            cy = unif1(generator);
        }
        double cr = sqrt(cx * cx + cy * cy + cz * cz);
        double cv = sqrt(6.674e-11 * 100000000000 / cr);
        double ca = atan2(cy, cx)+M_PI_2;
        double vx = cv * cos(ca);
        double vy = cv * sin(ca);
        double vz = 0;
        bodies.emplace_back(Body{ cx, cy, cz, vx, vy, vz, m, make_float3(100, 100, 100), make_float3(0, 0, 0), make_float3(0, 0, 0), 0, make_float2(0, 0), 1 });
    }
    NBody* bn = NULL;
    hipError_t err = NBody::makeNBody(bn, bodies, rad);
    Mat pimg = img.clone();

    Render r;
    r.init_render(bodies, make_float3(0, 0, 10000), make_float3(-1000, 0, 0), make_float3(0, -1000, 0), 10, 1000);

    for (int i = 0; i < 1000000000; i++) {
        img = Mat::zeros(1000, 1000, CV_8UC3);
        err = bn->step(1000, 10, 1024, 0);
        memcpy(&px, &x, sizeof(x));
        memcpy(&py, &y, sizeof(x));
        memcpy(&pz, &z, sizeof(x));
        err = bn->getPos(x, y, z);
        /*for (int j = 0; j < 100; j++) {
            cout<<"j: "<<j<<" dx: "<<x[j]-px[j]<<" dy: "<<y[j] - py[j] <<" dz: "<<z[j] - pz[j] <<endl;
        }*/
        //cout << "Error Pos: " << hipGetErrorString(err) << endl;
        uchar3 *img_res = new uchar3[1000000];
        err = r.render(img_res, 1000, 1000, 10, x, y, z, 10, 1024, 0);

        cout << "render status " << err << endl;
        for (int j = 0; j < 1000; j++) {
            for (int k = 0; k < 1000; k++) {
                int idx = j * 1000 + k;
                img.at<uchar3>(j, k) = img_res[idx];
            }
        }
        Mat fimg = img;
        putText(fimg, "Frame: " + to_string(i), Point(10, 30), FONT_HERSHEY_SIMPLEX, 1, Scalar(255, 255, 255), 2);
        imshow("Test", fimg);
        waitKey(1);
        delete [] img_res;
	}
    r.delete_render();

    delete[] x;
    delete[] y;
    delete[] z;
    delete[] px;
    delete[] py;
    delete[] pz;

    return 0;
}