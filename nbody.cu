#include "hip/hip_runtime.h"
#include "nbody.cuh"

using namespace std;


//verlet integration (position and velocity tick separately)
//to get around the limited precision of doubles, we will accumulate the changes until they are large enough to be significant 
//(the errors are most significant when small and large numbers are added and subtracted together), then push them to the main arrays
static __global__ void tickp(size_t n, double deltat, double* p, double* pt, double* v, double* a, bool *bounds, double radius) {
	int idx = threadIdx.x;
	int srd = blockDim.x;
	for (int i = idx; i < n; i += srd) {
		if (!bounds[i]) continue; //if the body is out of bounds, skip it
		pt[i] = pt[i] + v[i] * deltat + 0.5 * a[i] * deltat * deltat;
		if (pt[i] + p[i] != p[i]) {
			p[i] = pt[i] + p[i];
			pt[i] = 0;
		}
		if (fabs(p[i]) > radius) bounds[i] = false; //if the body is out of bounds, mark it as such
	}
}
static __global__ void tickv(size_t n, double deltat, double* v, double* vt, double* ao, double* an, bool* bounds) {
	int idx = threadIdx.x;
	int srd = blockDim.x;
	for (int i = idx; i < n; i += srd) {
		if (!bounds[i]) continue; //if the body is out of bounds, skip it
		vt[i] = vt[i] + 0.5 * (ao[i] + an[i]) * deltat;
		if (vt[i] + v[i] != v[i]) {
			v[i] = vt[i] + v[i];
			vt[i] = 0;
		}
	}
}

__global__ void updateAccel(size_t n, double g, double *x, double *y, double *z, double *ax, double *ay, double *az, double *m, bool *ib, double soft) {
	//now we calculate the new acceleration
	int idx = threadIdx.x;
	int srd = blockDim.x;
	for (int i = idx; i < n; i += srd) {
		if (!ib[i]) continue; //if the body is out of bounds, skip it
		ax[i] = 0;
		ay[i] = 0;
		az[i] = 0;
		for (int j = 0; j < n; j++) {
			if (!ib[j]) continue; //if the body is out of bounds, skip it
			if (i != j) {
				double dx = x[j] - x[i];
				double dy = y[j] - y[i];
				double dz = z[j] - z[i];
				double rsq = dx * dx + dy * dy + dz * dz;
				double rth = (rsq+soft) * sqrt(rsq+soft);
				double a = g * m[j] / rth;
				ax[i] += a * dx; //because there is a r vector on top of the equation usually, no need to divide by r
				ay[i] += a * dy;
				az[i] += a * dz;
			}
		}
	}
}
// making v const is theoretically the best practice, but it interferes with the memory copying process
hipError_t NBody::makeNBody(NBody*& nb, std::vector<Body>& v, double r) {
	nb = new NBody(r);
	nb->n = v.size();
	hipError_t err;

	// Concisely allocate and handle potential errors for all device arrays
	auto allocateAndCopy = [&](double*& deviceArray, const double* hostArray) -> hipError_t {
		err = hipMallocManaged(&deviceArray, nb->n * sizeof(double));
		if (err != hipSuccess) return err;
		return hipMemcpy(deviceArray, hostArray, nb->n * sizeof(double), hipMemcpyHostToDevice);
		};

	// Helper lambda for bool arrays
	auto allocateAndCopyBool = [&](bool*& deviceArray, const bool* hostArray) -> hipError_t {
		err = hipMallocManaged(&deviceArray, nb->n * sizeof(bool));
		if (err != hipSuccess) return err;
		return hipMemcpy(deviceArray, hostArray, nb->n * sizeof(bool), hipMemcpyHostToDevice);
		};

	// Prepare host arrays, for some reason vectors do not work well with the hipMemcpy function
	double *tx = new double[nb->n], *ty = new double[nb->n], *tz = new double[nb->n], *tvx = new double[nb->n], *tvy = new double[nb->n], *tvz = new double[nb->n], *tm = new double[nb->n];
	bool* tib = new bool[v.size()];  // Initialize all elements to true
	fill(tib, tib + nb->n, true);
	for (int i = 0; i < nb->n; i++) {
		tx[i] = v[i].x; ty[i] = v[i].y; tz[i] = v[i].z;
		tvx[i] = v[i].vx; tvy[i] = v[i].vy; tvz[i] = v[i].vz;
		tm[i] = v[i].m;
	}

	// allocate and copy arrays with error checking
	err = allocateAndCopy(nb->x, tx);
	if (err != hipSuccess) return err;
	err = allocateAndCopy(nb->y, ty);
	if (err != hipSuccess) return err;
	err = allocateAndCopy(nb->z, tz);
	if (err != hipSuccess) return err;
	err = allocateAndCopy(nb->vx, tvx);
	if (err != hipSuccess) return err;
	err = allocateAndCopy(nb->vy, tvy);
	if (err != hipSuccess) return err;
	err = allocateAndCopy(nb->vz, tvz);
	if (err != hipSuccess) return err;
	err = allocateAndCopy(nb->m, tm);
	if (err != hipSuccess) return err;
	err = allocateAndCopyBool(nb->inbounds, tib);
	if (err != hipSuccess) return err;

	delete[] tx, ty, tz, tvx, tvy, tvz, tm, tib;

	// Allocate acceleration arrays
	err = hipMallocManaged(&nb->axo, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->ayo, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->azo, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->axn, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->ayn, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->azn, v.size() * sizeof(double));
	if (err != hipSuccess) return err;

	err = hipMallocManaged(&nb->xt, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->yt, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->zt, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->vxt, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->vyt, v.size() * sizeof(double));
	if (err != hipSuccess) return err;
	err = hipMallocManaged(&nb->vzt, v.size() * sizeof(double));
	if (err != hipSuccess) return err;

	updateAccel << <1, 1 >> > (nb->n, nb->g, nb->x, nb->y, nb->z, nb->axn, nb->ayn, nb->azn, nb->m, nb->inbounds, 1); //populate the initial acceleration arrays, "ease in" with larger softening
	hipDeviceSynchronize();
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}

	return hipSuccess;
}


hipError_t NBody::destroyNBody(NBody* nb)
{
	hipError_t err = hipFree(nb->x);
	if (err != hipSuccess) return err;
	err = hipFree(nb->y);
	if (err != hipSuccess) return err;
	err = hipFree(nb->z);
	if (err != hipSuccess) return err;

	err = hipFree(nb->vx);
	if (err != hipSuccess) return err;
	err = hipFree(nb->vy);
	if (err != hipSuccess) return err;
	err = hipFree(nb->vz);
	if (err != hipSuccess) return err;

	err = hipFree(nb->xt);
	if (err != hipSuccess) return err;
	err = hipFree(nb->yt);
	if (err != hipSuccess) return err;
	err = hipFree(nb->zt);
	if (err != hipSuccess) return err;
	err = hipFree(nb->vxt);
	if (err != hipSuccess) return err;
	err = hipFree(nb->vyt);
	if (err != hipSuccess) return err;
	err = hipFree(nb->vzt);
	if (err != hipSuccess) return err;


	err = hipFree(nb->axo);
	if (err != hipSuccess) return err;
	err = hipFree(nb->ayo);
	if (err != hipSuccess) return err;
	err = hipFree(nb->azo);
	if (err != hipSuccess) return err;
	err = hipFree(nb->axn);
	if (err != hipSuccess) return err;
	err = hipFree(nb->ayn);
	if (err != hipSuccess) return err;
	err = hipFree(nb->azn);
	if (err != hipSuccess) return err;

	err = hipFree(nb->m);
	if (err != hipSuccess) return err;

	err = hipFree(nb->inbounds);
	if (err != hipSuccess) return err;

	delete nb;

	return hipSuccess;
}

//full verlet integration step, will return error code if something goes wrong
hipError_t NBody::step(double deltat, size_t nblocks, int tpb, int device) {
	hipSetDevice(device);
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	tickp<<<nblocks, tpb>>>(n, deltat, x, xt, vx, axn, inbounds, radius);
	hipDeviceSynchronize();
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	tickp<<<nblocks, tpb>>>(n, deltat, y, yt, vy, ayn, inbounds, radius);
	hipDeviceSynchronize();
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	tickp<<<nblocks, tpb>>>(n, deltat, z, zt, vz, azn, inbounds, radius);
	hipDeviceSynchronize();
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	swap(axo, axn); //prep the acceleration arrays for update by shuffling addresses
	swap(ayo, ayn);
	swap(azo, azn);
	updateAccel<<<nblocks, tpb>>>(this->n, this->g, this->x, this->y, this->z, this->axn, this->ayn, this->azn, this->m, this->inbounds, this->softening);
	hipDeviceSynchronize();
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	tickv<<<nblocks, tpb>>>(n, deltat, vx, vxt, axo, axn, inbounds);
	hipDeviceSynchronize();
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	tickv<<<nblocks, tpb>>>(n, deltat, vy, vyt, ayo, ayn, inbounds);
	hipDeviceSynchronize();
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	tickv<<<nblocks, tpb>>>(n, deltat, vz, vzt, azo, azn, inbounds);
	hipDeviceSynchronize();
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}

	return hipSuccess;

}
hipError_t  NBody::getPos(double* x, double* y, double* z) {
	hipMemcpy(x, this->x, n * sizeof(double), hipMemcpyDeviceToHost);
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	hipMemcpy(y, this->y, n * sizeof(double), hipMemcpyDeviceToHost);
	if (hipPeekAtLastError() - hipSuccess) {
		return hipPeekAtLastError();
	}
	hipMemcpy(z, this->z, n * sizeof(double), hipMemcpyDeviceToHost);
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	return hipSuccess;
}
hipError_t  NBody::getVel(double* vx, double* vy, double* vz) {
	hipMemcpy(vx, this->vx, n * sizeof(double), hipMemcpyDeviceToHost);
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	hipMemcpy(vy, this->vy, n * sizeof(double), hipMemcpyDeviceToHost);
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	hipMemcpy(vz, this->vz, n * sizeof(double), hipMemcpyDeviceToHost);
	if (hipPeekAtLastError() != hipSuccess) {
		return hipPeekAtLastError();
	}
	return hipSuccess;
}


