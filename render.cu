#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include "render.cuh"

//helper implementations
__device__ float3 operator+(const float3 &a, const float3 &b) {
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}
__device__ float3 operator-(const float3 &a, const float3 &b) {
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
__device__ double dot(float3 a, float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}
__device__ float3 operator*(const float3 &a, const double &b) {
	return make_float3(a.x * b, a.y * b, a.z * b);
}
__device__ double distance(const float3 &a, const float3 &b) {
	return sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y) + (a.z - b.z) * (a.z - b.z));
}
__device__ float3 normalize(const float3 &a) {
	double mag = sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
	return make_float3(a.x / mag, a.y / mag, a.z / mag);
}
__device__ double angle(const float3 &a, const float3 &b) {
	return acos(dot(a, b) / (sqrt(dot(a, a)) * sqrt(dot(b, b))));
}

//calculation functions for determining ray intersections
__device__ void vec_point_closest(float3& result, float3& origin, float3& direction, float3& point) {
	float3 v = point - origin;
	result = v - (direction * dot(v, direction)) + point;
} //this function calculates the closest point on a line to a given point
__device__ void vec_sphere_intersect(bool& intersects, float3& result, float3& origin, float3& direction, float3& center, double radius) {
	double a = dot(direction, direction);
	double b = 2 * dot(direction, origin - center);
	double c = dot(origin - center, origin - center) - radius * radius;
	double discriminant = b * b - 4 * a * c;
	if (discriminant <= 0) { //if the ray either misses or skims the sphere, there is no need to apply any transforms to it
		intersects = false;
		return;
	}
	double t1 = (-b + sqrt(discriminant)) / (2 * a);
	double t2 = (-b - sqrt(discriminant)) / (2 * a);
	if(t1 < 0 && t2 < 0) { //ray never intersects
		intersects = false;
		return;
	}
	else if(t1 < 0) { //ray intersects at t2 only
		intersects = true;
		result = origin + direction * t2;
		return;
	}
	else if (t2 < 0) { //ray intersects at t1 only
		intersects = true;
		result = origin + direction * t1;
		return;
	}
	intersects = true;
	double dist = fmin(t1, t2);
	result = origin + direction * dist;
} //this function calculates the intersection of a ray with a sphere, if it exists
__device__ void recolor(float3& rgb, float3& rat) {
	rgb.x *= rat.x;
	rgb.y *= rat.y;
	rgb.z *= rat.z;
} //this function applies the color transformation to a ray
__device__ void shuffle(int *array, size_t n, hiprandState state) {
	if (n > 1) {
		size_t i;
		for (i = 0; i < n - 1; i++) {
			size_t j = i + hiprand_uniform_double(&state) * (n - i);
			int t = array[j];
			array[j] = array[i];
			array[i] = t;
		}
	}
} //this function shuffles an array

static __global__ void render_kernel(int rn, int bn, //number of rays and bodies
	float3* ro, float3 *rd, float3 *rgb, bool *active, // ray arrays
	float3* bpos, float2* brad, float3* irat, float3* orat, double* trat, double *m, double *eh, double *ref_idx, int *last_bounce, int *shuffled) { // body info
	const double g = 6.67430e-11;
	const double v_light = 299792458.0;

	int idx = threadIdx.x;
	int srd = blockDim.x;
	int tId = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState state;
	hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);
	for (int i = idx; i < rn; i += srd) {
		if(!active[i]) continue;
		shuffle(shuffled, bn, state);
		for (int k = 0; k < bn; k++) {
			int j = shuffled[k]; //we randomize the order of the bodies to avoid bias in the calculations
			if(j == last_bounce[i]) continue; //the ray just bounced off this body!
			if(!active[i]) continue;
			float3 closest_point;
			vec_point_closest(closest_point, ro[i], rd[i], bpos[j]);
			if (distance(bpos[j], closest_point) < eh[j]) {
				rgb[i] = make_float3(0, 0, 0); //absorption
				active[i] = false;
			}
			float3 outer_intersection;
			bool outer_intersects;
			vec_sphere_intersect(outer_intersects, outer_intersection, ro[i], rd[i], bpos[j], brad[j].y);
			//outer (refractive) sphere logic
			//for simplicity, we will assume uniform refraction index
			if (!outer_intersects) { //gravitational lensing is usually minor in comparison to other effects, so we only apply it if the ray misses the outer sphere entirely
				double dist = distance(closest_point, bpos[j]);
				double alpha = 4*g*m[j]/(v_light*v_light*dist);
				float3 deflection_norm = normalize(bpos[j] - closest_point);
				last_bounce[i] = j;
				ro[i] = closest_point;
				rd[i] = normalize(rd[i] + deflection_norm * tan(alpha));
				continue;
			}
			else {
				float3 refraction_norm = normalize(bpos[j] - outer_intersection); //this points inwards as it is more convenient for the calculations
				float3 deflect = rd[i] - refraction_norm * dot(rd[i], refraction_norm);
				double outer_angle = angle(rd[i], refraction_norm);
				float3 forward = normalize((refraction_norm * dot(rd[i], refraction_norm)) + (deflect * (tan(outer_angle/ref_idx[j]) / tan(outer_angle))));
				bool reflected = trat[j] < hiprand_uniform_double(&state);
				bool fw_inner_intersects;
				float3 fw_inner_intersection;
				vec_sphere_intersect(fw_inner_intersects, fw_inner_intersection, outer_intersection, forward, bpos[j], brad[j].x);
				float3 start;
				if (fw_inner_intersects) {
					double where = hiprand_uniform_double(&state);
					start = outer_intersection * where + fw_inner_intersection * (1 - where);
				} else {
					float3 closest_point;
					vec_point_closest(closest_point, outer_intersection, forward, bpos[j]);
					double where = hiprand_uniform_double(&state);
					start = outer_intersection * where + fw_inner_intersection * (1 - where);
				}
				// lambertian scattering, slightly modified
				float3 x_offset = make_float3(1, 0, 0) * (hiprand_uniform_double(&state) - 0.5);
				float3 y_offset = make_float3(0, 1, 0) * (hiprand_uniform_double(&state) - 0.5);
				float3 z_offset = make_float3(0, 0, 1) * (hiprand_uniform_double(&state) - 0.5);
				float3 scatter_radius_vector = normalize(x_offset + y_offset + z_offset);
				float3 scatter_direction = reflected? (make_float3(0, 0, 0) - forward) : forward;
				ro[i] = start;
				rd[i] = normalize(scatter_direction + scatter_radius_vector);
				recolor(rgb[i], orat[j]);
			}
			float3 inner_intersection;
			bool inner_intersects;
			vec_sphere_intersect(inner_intersects, inner_intersection, ro[i], rd[i], bpos[j], brad[j].x);
			//inner (reflective) sphere logic
			if (inner_intersects) {
				float3 x_offset = make_float3(1, 0, 0) * (hiprand_uniform_double(&state) - 0.5);
				float3 y_offset = make_float3(0, 1, 0) * (hiprand_uniform_double(&state) - 0.5);
				float3 z_offset = make_float3(0, 0, 1) * (hiprand_uniform_double(&state) - 0.5);
				float3 normal = normalize(inner_intersection - bpos[j]);
				float3 scatter_radius_vector = normalize(x_offset + y_offset + z_offset);
				ro[i] = inner_intersection;
				rd[i] = normalize(normal + scatter_radius_vector);
				recolor(rgb[i], irat[j]);
				vec_sphere_intersect(inner_intersects, inner_intersection, ro[i], rd[i], bpos[j], brad[j].y);
				// if the ray started inside the outer sphere, it has to intersect with it once
				//inner intersection is now used as a temp variable
				//snell's law to calculate the "forward" direction
				float3 refraction_norm = normalize(outer_intersection - bpos[j]); //this points outwards this time
				float3 deflect = rd[i] - refraction_norm * dot(rd[i], refraction_norm);
				double outer_angle = angle(rd[i], refraction_norm);
				float3 forward = normalize((refraction_norm * dot(rd[i], refraction_norm)) + (deflect * (tan(outer_angle) / tan(outer_angle / ref_idx[j]))));
				//origin
				double where = hiprand_uniform_double(&state);
				ro[i] = ro[i] * where + inner_intersection * (1 - where);
				x_offset = make_float3(1, 0, 0) * (hiprand_uniform_double(&state) - 0.5);
				y_offset = make_float3(0, 1, 0) * (hiprand_uniform_double(&state) - 0.5);
				z_offset = make_float3(0, 0, 1) * (hiprand_uniform_double(&state) - 0.5);
				scatter_radius_vector = normalize(x_offset + y_offset + z_offset);
				rd[i] = normalize(forward + scatter_radius_vector);
			}
			//get the origin out of the sphere to avoid double intersections
			//this is done by moving the origin to the intersection point and then moving it a bit further
			vec_sphere_intersect(inner_intersects, inner_intersection, ro[i], rd[i], bpos[j], brad[j].y);
			if (inner_intersects) {
				ro[i] = inner_intersection + rd[i] * 0.0001;
			}
			last_bounce[i] = j;
		}
	}
}

__global__ void render_init(int nbodies, int rays_per, float3* bodies_pos, float3* bodies_rgb, float2* bodies_rad, float3* origins, float3* directions, float3* rgb, bool* active, int* last_bounce) {
	int idx = threadIdx.x;
	int srd = blockDim.x;
	int tId = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState state;
	hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);
	for (int i = idx; i < nbodies; i += srd) {
		for (int j = 0; j < rays_per; j++) {
			float3 x_offset = make_float3(1, 0, 0) * (hiprand_uniform_double(&state) - 0.5);
			float3 y_offset = make_float3(0, 1, 0) * (hiprand_uniform_double(&state) - 0.5);
			float3 z_offset = make_float3(0, 0, 1) * (hiprand_uniform_double(&state) - 0.5);
			origins[i * rays_per + j] = bodies_pos[i];
			directions[i * rays_per + j] = normalize(x_offset + y_offset + z_offset);
			rgb[i * rays_per + j] = bodies_rgb[i];
			active[i * rays_per + j] = true;
			last_bounce[i * rays_per + j] = -1;
			bool tii;
			float3 ti;
			//move ray out of the body sphere
			vec_sphere_intersect(tii, ti, origins[i], directions[i], bodies_pos[j], bodies_rad[j].y);
			origins[i] = ti + directions[i] * 0.0001;
		}
	}
}

double dot_nodevice(float3 a, float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}
double angle_nodevice(float3 a, float3 b) {
	return acos(dot_nodevice(a, b) / (sqrt(dot_nodevice(a, a)) * sqrt(dot_nodevice(b, b))));
}
float3 prod_nodevice(float3 a, double b) {
	return make_float3(a.x * b, a.y * b, a.z * b);
}
float3 sum_nodevice(initializer_list<float3> a) {
	float3 result = make_float3(0, 0, 0);
	for (auto i : a) {
		result.x += i.x;
		result.y += i.y;
		result.z += i.z;
	}
	return result;
}
float3 diff_nodevice(float3 a, float3 b) {
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
double veclen_nodevice(float3 a) {
	return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}
float3 norm_nodevice(float3 a) {
	double mag = sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
	return make_float3(a.x / mag, a.y / mag, a.z / mag);
}
float3 cross_nodevice(float3 a, float3 b) {
	return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

hipError_t Render::render(int3* image_res, int width, int height, int max_bounces, double* x, double* y, double *z, size_t nblocks, int tpb, int device) {
	hipSetDevice(device);
	float3* bpos = new float3[nbodies];
	for (int i = 0; i < nbodies; i++) {
		bpos[i] = make_float3(x[i], y[i], z[i]);
	}
	hipMemcpy(bodies_pos, bpos, sizeof(float3) * nbodies, hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) {
		return hipGetLastError();
	}
	render_init<<<nblocks, tpb>>>(nbodies, rays_per, bodies_pos, bodies_rgb, bodies_rad, origins, directions, rgb, active, last_bounce);
	for (int i = 0; i < max_bounces; i++) {
		render_kernel << <nblocks, tpb >> > (nbodies*rays_per, nbodies, origins, directions, rgb, active, bodies_pos, bodies_rad, inner_ratios, outer_ratios, transl_ratios, mass, event_horizon, ref_idx, last_bounce, shuffled); 
		if (hipGetLastError() != hipSuccess) {
			return hipGetLastError();
		}
		hipDeviceSynchronize();
		if (hipGetLastError() != hipSuccess) {
			return hipGetLastError();
		}
	}
	float3 *t_origins = new float3[nbodies * rays_per];
	float3 *t_directions = new float3[nbodies * rays_per];
	float3 *t_rgb = new float3[nbodies * rays_per];
	bool *t_active = new bool[nbodies * rays_per];
	hipMemcpy(t_origins, origins, sizeof(float3) * nbodies * rays_per, hipMemcpyDeviceToHost);
	if (hipGetLastError() != hipSuccess) {
		return hipGetLastError();
	}
	hipMemcpy(t_directions, directions, sizeof(float3) * nbodies * rays_per, hipMemcpyDeviceToHost);
	if (hipGetLastError() != hipSuccess) {
		return hipGetLastError();
	}
	hipMemcpy(t_rgb, rgb, sizeof(float3) * nbodies * rays_per, hipMemcpyDeviceToHost);
	if (hipGetLastError() != hipSuccess) {
		return hipGetLastError();
	}
	hipMemcpy(t_active, active, sizeof(bool) * nbodies * rays_per, hipMemcpyDeviceToHost);
	if (hipGetLastError() != hipSuccess) {
		return hipGetLastError();
	}
	float3 *image = new float3[width * height];
	for(int i = 0; i < width*height; i++) {
		image[i] = make_float3(0, 0, 0);
	}
	//put the rays onto the image
	for (int i = 0; i < nbodies * rays_per; i++) {
		if(!t_active[i]) continue;
		double angleToNormal = angle_nodevice(t_directions[i], cam_norm);
		if (angleToNormal > M_PI / 2) {
			continue;
		}
		float3 diff = diff_nodevice(t_origins[i], cam_bl);
		double prod1 = dot_nodevice(diff, cam_norm);
		double prod2 = dot_nodevice(t_directions[i], cam_norm);
		double prod3 = prod1 / prod2;
		float3 hit = diff_nodevice(t_origins[i], prod_nodevice(t_directions[i], prod3));
		double hd = dot_nodevice(diff_nodevice(hit, cam_bl), cam_h) / veclen_nodevice(cam_h);
		double vd = dot_nodevice(diff_nodevice(hit, cam_bl), cam_v) / veclen_nodevice(cam_v);
		if(hd < 0 || hd > veclen_nodevice(cam_h) || vd < 0 || vd > veclen_nodevice(cam_v)) { //outside the camera screen!
			continue;
		}
		float3 image_focus = sum_nodevice({ cam_bl, prod_nodevice(cam_h, 0.5), prod_nodevice(cam_v, 0.5), prod_nodevice(cam_norm, focal_length) });
		float3 image_diff_norm = norm_nodevice(diff_nodevice(image_focus, hit));
		int x = (int)(width*hd / veclen_nodevice(cam_h));
		int y = (int)(height*vd / veclen_nodevice(cam_v));
		image[y*width + x] = sum_nodevice({image[y*width + x], prod_nodevice(t_rgb[i], dot_nodevice(t_directions[i], image_diff_norm))});
	}
	//map the image to standard 0-255 range
	double abs_max_color = 0;
	for (int i = 0; i < width * height; i++) {
		abs_max_color = max(abs_max_color, (double)(image[i].x));
		abs_max_color = max(abs_max_color, (double)(image[i].y));
		abs_max_color = max(abs_max_color, (double)(image[i].z));
	}
	for(int i = 0; i < width*height; i++) {
		image_res[i].x = (int)(image[i].x * 255 / abs_max_color);
		image_res[i].y = (int)(image[i].y * 255 / abs_max_color);
		image_res[i].z = (int)(image[i].z * 255 / abs_max_color);
	}
	return hipSuccess;
}

hipError_t Render::init_render(vector<Body> bodies, float3 cbl, float3 ch, float3 cv, int rays_per, double fl) {
	hipMallocManaged(&bodies_pos, sizeof(float3) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&bodies_rgb, sizeof(float3) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&bodies_rad, sizeof(float2) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&inner_ratios, sizeof(float3) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&outer_ratios, sizeof(float3) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&transl_ratios, sizeof(double) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&mass, sizeof(double) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&event_horizon, sizeof(double) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&ref_idx, sizeof(double) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&origins, sizeof(float3) * bodies.size() * rays_per);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&directions, sizeof(float3) * bodies.size() * rays_per);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&rgb, sizeof(float3) * bodies.size() * rays_per);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&active, sizeof(bool) * bodies.size() * rays_per);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&last_bounce, sizeof(int) * bodies.size() * rays_per);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMallocManaged(&shuffled, sizeof(int) * bodies.size());
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	nbodies = bodies.size();
	this->rays_per = rays_per;
	cam_bl = cbl;
	cam_h = ch;
	cam_v = cv;
	cam_norm = norm_nodevice(cross_nodevice(cam_h, cam_v));
	focal_length = fl;
	float3* bpos = new float3[nbodies];
	float3* brgb = new float3[nbodies];
	float2* brad = new float2[nbodies];
	float3* irat = new float3[nbodies];
	float3* orat = new float3[nbodies];
	double* trat = new double[nbodies];
	double* m = new double[nbodies];
	double* eh = new double[nbodies];
	double* ri = new double[nbodies];
	const double g = 6.67430e-11;
	const double v_light = 299792458.0;
	for (int i = 0; i < nbodies; i++) {
		bpos[i] = make_float3(bodies[i].x, bodies[i].y, bodies[i].z);
		brgb[i] = bodies[i].rgb;
		brad[i] = bodies[i].radius;
		irat[i] = bodies[i].inner_ratio;
		orat[i] = bodies[i].outer_ratio;
		trat[i] = bodies[i].transl_ratio;
		m[i] = bodies[i].m;
		eh[i] = 2 * g * m[i] / (v_light * v_light); //schwarzschild radius
		ri[i] = bodies[i].ref_idx;
	}
	hipMemcpy(bodies_pos, bpos, sizeof(float3) * nbodies, hipMemcpyHostToDevice);
	if(hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMemcpy(bodies_rgb, brgb, sizeof(float3) * nbodies, hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMemcpy(bodies_rad, brad, sizeof(float2) * nbodies, hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMemcpy(inner_ratios, irat, sizeof(float3) * nbodies, hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMemcpy(outer_ratios, orat, sizeof(float3) * nbodies, hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMemcpy(transl_ratios, trat, sizeof(double) * nbodies, hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMemcpy(mass, m, sizeof(double) * nbodies, hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMemcpy(event_horizon, eh, sizeof(double) * nbodies, hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipMemcpy(ref_idx, ri, sizeof(double) * nbodies, hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	return hipSuccess;
}
hipError_t Render::delete_render() {
	hipFree(bodies_pos);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(bodies_rgb);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(bodies_rad);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(inner_ratios);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(outer_ratios);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(transl_ratios);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(mass);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(event_horizon);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(ref_idx);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(origins);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(directions);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(rgb);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(active);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(last_bounce);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();
	hipFree(shuffled);
	if (hipGetLastError() != hipSuccess) return hipGetLastError();

	return hipSuccess;
}

